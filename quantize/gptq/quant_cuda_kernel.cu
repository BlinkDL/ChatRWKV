#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// atomicAdd for double-precision floating-point numbers on hardware with
// compute capability < 6.0 from:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(
    double* address,
    double val
) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(
      address_as_ull,
      assumed,
      __double_as_longlong(val + __longlong_as_double(assumed))
    );

  // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height, 	
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height, 	
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height, 	
    int height,
    int width,
	int zero_width
);

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
	const  	    int* __restrict__ g_idx,
    int batch,
    int vec_height, 	
    int height,
    int width,
	int zero_width
);

const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT2 =  16;
const int BLOCKHEIGHT3 =  24;
const int BLOCKHEIGHT4 =  32;
const int BLOCKHEIGHT8 =  64;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}


void vecquant2matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant2matmul_cuda", ([&] {
      VecQuant2MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(), 
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  
  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 16;
  int k;
  unsigned int g;
  scalar_t w_tmp;
  
  int z_w = w / 16; 
  int z_mod = (w % 16) * 2;
  
  float weight[BLOCKWIDTH];
  
  for (k = 0; k <  BLOCKWIDTH; ++k){	
	int k_w = (k / 16); 
	int k_bit = (k % 16) * 2;
	
    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scalar_t((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod & 0x3) + 1);
	
    w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0x3);
    
	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){	
	res = 0;
	
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){	
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
  }
}

void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda", ([&] {
      VecQuant3MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(), 
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  
  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = (h / 3) * 32;
  int k;
  unsigned int g;
  scalar_t w_tmp;
  
  int z_w = (w / 32) * 3; 
  int z_mod = w % 32;
  int z_bit;
  unsigned int z_tmp;
  if (z_mod != 10){
    if (z_mod != 21){
      z_bit = z_mod;
      if (z_bit > 21){
        z_bit -= 22;
        z_bit *= 3;
        z_bit += 2;
        z_w += 2;
      } else if (z_bit > 10){
        z_bit -= 11;
        z_bit *= 3;
        z_bit += 1;
        z_w += 1;
      } else {
        z_bit *= 3;
      }
    } else {
      z_w += 1;
    }
  }
  
  float weight[BLOCKWIDTH];
  
  for (k = 0; k <  BLOCKWIDTH; ++k){	
	int k_w = (k / 32) * 3; 
	int k_mod = k % 32;
	int k_bit;
	  
	if (k_mod != 10){
	  if (k_mod != 21){
        k_bit = k_mod;
        if (k_bit > 21){
		  k_bit -= 22;
		  k_bit *= 3;
		  k_bit += 2;
		  k_w += 2;
        } else if (k_bit > 10){
		  k_bit -= 11;
		  k_bit *= 3;
		  k_bit += 1;
		  k_w += 1;
        } else {
		  k_bit *= 3;
        }
	  } else {
        k_w += 1;
	  }
	}
	
    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero;
    if (z_mod == 10) {
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 30) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 2) & 0x4);
      zero = scalar_t((z_tmp) + 1);
    } else if (z_mod == 21){
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 31) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 1) & 0x6);
      zero = scalar_t((z_tmp) + 1);
    } else {
      zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_bit) & 0x7) + 1);
    }
	
    if (k_mod == 10) {
      w_tmp = (as_unsigned(mat[i + (k_w * width)]) >> 30) | ((as_unsigned(mat[i + ((k_w + 1)* width)]) << 2) & 0x4);
    } else if (k_mod == 21){
      w_tmp = (as_unsigned(mat[i + (k_w * width)]) >> 31) | ((as_unsigned(mat[i + ((k_w + 1)* width)]) << 1) & 0x6);
    } else {
      w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0x7);
    }
	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){	
	res = 0;
	
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){	
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
  }
}

void vecquant4matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_cuda", ([&] {
      VecQuant4MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(), 
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  
  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 8;
  int k;
  unsigned int g;
  scalar_t w_tmp;
  

  int z_w = w / 8; 
  int z_mod = (w % 8) * 4;
  
  float weight[BLOCKWIDTH];
  
  for (k = 0; k <  BLOCKWIDTH; ++k){	
	int k_w = (k / 8); 
	int k_bit = (k % 8) * 4;
	
    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1);
	
    w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0xF);
    
	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){	
	res = 0;
	
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){	
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
  }
}

void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  torch::Tensor g_idx
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(), g_idx.data<int>(), 
        batch, vec_height, height, width, zero_width
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    const   	int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
	int zero_width
) {
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  
  __shared__ scalar_t blockvec[BLOCKWIDTH];
  int i = width * h + w;
  int g_h = h * 4;
  int k;
  unsigned int g;
  scalar_t w_tmp;
  
  int z_w = w / 4; 
  int z_mod = (w % 4) * 8;
  
  float weight[BLOCKWIDTH];
  
  for (k = 0; k <  BLOCKWIDTH; ++k){	
	int k_w = (k / 4); 
	int k_bit = (k % 4) * 8;
	
    g = as_int(g_idx[g_h + k]);
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xFF) + 1);
	
    w_tmp = ((as_unsigned(mat[i + (k_w * width)]) >> k_bit) & 0xFF);
    
	weight[k] = scale * (w_tmp - zero);
  }

  scalar_t res;
  for (int b = 0; b < batch; ++b){	
	res = 0;
	
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
    __syncthreads();
	for (k = 0; k <  BLOCKWIDTH; ++k){	
	  res += weight[k] * blockvec[k];
    }
    atomicAdd(&mul[b * width + w], res);
  }
}
