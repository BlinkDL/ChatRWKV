#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"

typedef at::Half fp16;
typedef at::BFloat16 bf16;
typedef float fp32;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               float *__restrict__ _state, const F *__restrict__ const _r, const F *__restrict__ const _w, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const _a, const F *__restrict__ const _b,
                               F *__restrict__ const _y)
{
    const int e = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _state += h*_N_*_N_ + i*_N_; // wrong if B > 1 !!!

    float state[_N_];
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        state[j] = _state[j];

    __shared__ float r[_N_], k[_N_], w[_N_], a[_N_], b[_N_];

    for (int _t = 0; _t < T; _t++)
    {
        const int t = e*T*C + h*_N_ + i + _t * C;
        __syncthreads();
        r[i] = float(_r[t]);
        w[i] = __expf(-__expf(float(_w[t])));
        k[i] = float(_k[t]);
        a[i] = float(_a[t]);
        b[i] = float(_b[t]);
        __syncthreads();

        float sa = 0;
        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            sa += a[j] * state[j];
        }

        float vv = float(_v[t]);
        float y = 0;
        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            s = s * w[j] + k[j] * vv + sa * b[j];
            y += s * r[j];
        }
        _y[t] = F(y);
    }
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        _state[j] = state[j];    
}

void cuda_forward_bf16(int B, int T, int C, int H, float *state, bf16 *r, bf16* w, bf16 *k, bf16 *v, bf16 *a, bf16 *b, bf16 *y)
{
    assert(H*_N_ == C);
    assert(B == 1); // only for B=1
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, w, k, v, a, b, y);
}
void cuda_forward_fp16(int B, int T, int C, int H, float *state, fp16 *r, fp16* w, fp16 *k, fp16 *v, fp16 *a, fp16 *b, fp16 *y)
{
    assert(H*_N_ == C);
    assert(B == 1); // only for B=1
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, w, k, v, a, b, y);
}
void cuda_forward_fp32(int B, int T, int C, int H, float *state, fp32 *r, fp32* w, fp32 *k, fp32 *v, fp32 *a, fp32 *b, fp32 *y)
{
    assert(H*_N_ == C);
    assert(B == 1); // only for B=1
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, w, k, v, a, b, y);
}
