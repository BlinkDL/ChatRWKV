#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "element_wise.h"
#include "util.h"

// Equivalent Python code:
// ww = t_first + k
// p = torch.maximum(pp, ww)
// e1 = torch.exp(pp - p)
// e2 = torch.exp(ww - p)
// wkv = ((e1 * aa + e2 * v) / (e1 * bb + e2)).to(dtype=x.dtype)
// ww = t_decay + pp
// p = torch.maximum(ww, k)
// e1 = torch.exp(ww - p)
// e2 = torch.exp(k - p)
// t1 = e1 * aa + e2 * v
// t2 = e1 * bb + e2
// r = r * wkv
// return t1, t2, p, r
struct WkvForwardOne {
  const float *t_first;
  const float *k;
  const float *pp;
  const float *aa;
  const float *bb;
  const float *t_decay;
  const float *v;
  /* out */ float *t1;
  /* out */ float *t2;
  /* out */ float *p;
  /* in & out */ half *r;

  __device__ void operator()(int i) const {
    float ww = t_first[i] + k[i];
    float pp_ = pp[i];
    float p_ = (pp_ > ww) ? pp_ : ww;
    float e1 = expf(pp_ - p_);
    float e2 = expf(ww - p_);
    float aa_ = aa[i];
    float bb_ = bb[i];
    float v_ = v[i];
    r[i] = __hmul(r[i], __float2half(((e1 * aa_ + e2 * v_) / (e1 * bb_ + e2))));
    ww = t_decay[i] + pp_;
    float k_ = k[i];
    p_ = (ww > k_) ? ww : k_;
    e1 = expf(ww - p_);
    e2 = expf(k_ - p_);
    t1[i] = e1 * aa_ + e2 * v_;
    t2[i] = e1 * bb_ + e2;
    p[i] = p_;
  }
};

/*
   Equivalent Python code:
   kx = xx * k_mix + sx * (1 - k_mix)
   vx = xx * v_mix + sx * (1 - v_mix)
   rx = xx * r_mix + sx * (1 - r_mix)
*/

struct Mix {
  const half *xx;
  const half *sx;
  const half *k_mix;
  const half *v_mix;
  const half *r_mix;
  /* out */ half *kx;
  /* out */ half *vx;
  /* out */ half *rx;

  __device__ void operator()(int i) const {
    half xx_ = xx[i];
    half sx_ = sx[i];
    half k_mix_ = k_mix[i];
    half v_mix_ = v_mix[i];
    half r_mix_ = r_mix[i];
    kx[i] = __hadd(__hmul(xx_, k_mix_),
                   __hmul(sx_, __hsub(__float2half(1), k_mix_)));
    vx[i] = __hadd(__hmul(xx_, v_mix_),
                   __hmul(sx_, __hsub(__float2half(1), v_mix_)));
    rx[i] = __hadd(__hmul(xx_, r_mix_),
                   __hmul(sx_, __hsub(__float2half(1), r_mix_)));
  }
};

using torch::Tensor;

void gemm_fp16_cublas_tensor(Tensor a, Tensor b, Tensor c);

Tensor att_one(Tensor x, Tensor ln_w, Tensor ln_b, Tensor sx, Tensor k_mix,
             Tensor v_mix, Tensor r_mix, Tensor kw,
             /* imm */ Tensor kx, Tensor vw, /* imm */ Tensor vx, Tensor rw,
             /* imm */ Tensor rx, Tensor ow, Tensor t_first,
             /* imm */ Tensor k, Tensor pp, Tensor ww, Tensor aa, Tensor bb,
             Tensor t_decay, /* imm */ Tensor v, /* in & out */ Tensor r,
             /* out */ Tensor x_plus_out, /* out */ Tensor t1,
             /* out */ Tensor t2, /* out */ Tensor p) {
  Tensor xx = at::layer_norm(x, {x.size(-1)}, ln_w, ln_b);
  element_wise(Mix{data_ptr<half>(xx), data_ptr<half>(sx),
                   data_ptr<half>(k_mix), data_ptr<half>(v_mix),
                   data_ptr<half>(r_mix), data_ptr<half>(kx),
                   data_ptr<half>(vx), data_ptr<half>(rx)},
               x.numel());

  gemm_fp16_cublas_tensor(kx, kw, k);
  gemm_fp16_cublas_tensor(vx, vw, v);
  gemm_fp16_cublas_tensor(rx, rw, r);
  at::sigmoid_(r);

  element_wise(WkvForwardOne{data_ptr<float>(t_first), data_ptr<float>(k),
                             data_ptr<float>(pp), data_ptr<float>(aa),
                             data_ptr<float>(bb), data_ptr<float>(t_decay),
                             data_ptr<float>(v), data_ptr<float>(t1),
                             data_ptr<float>(t2), data_ptr<float>(p),
                             data_ptr<half>(r)},
               x.numel());

  gemm_fp16_cublas_tensor(r, ow, x_plus_out);
  x_plus_out += x;
  return xx;
}
