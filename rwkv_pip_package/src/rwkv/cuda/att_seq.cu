#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "util.h"
#include "element_wise.h"

using torch::Tensor;

void gemm_fp16_cublas(const void *a, const void *b, void *c, int m,
                      int n, int k, bool output_fp32);

// based on `kernel_wkv_forward`, fusing more operations
__global__ void kernel_wkv_forward_new(
    const int B, const int T, const int C, const float *__restrict__ const _w,
    const float *__restrict__ const _u, const float *__restrict__ const _k,
    const float *__restrict__ const _v, const half *__restrict__ const r,
    half *__restrict__ const _y, float *__restrict__ const _aa,
    float *__restrict__ const _bb, float *__restrict__ const _pp) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int _b = idx / C;
  const int _c = idx % C;
  const int _offset = _b * T * C + _c;
  const int _state_offset = _b * C + _c;

  float u = _u[_c];
  float w = _w[_c];
  const float *__restrict__ const k = _k + _offset;
  const float *__restrict__ const v = _v + _offset;
  half *__restrict__ const y = _y + _offset;

  float aa = _aa[_state_offset];
  float bb = _bb[_state_offset];
  float pp = _pp[_state_offset];
  for (int i = 0; i < T; i++) {
    const int ii = i * C;
    const float kk = k[ii];
    const float vv = v[ii];
    float ww = u + kk;
    float p = max(pp, ww);
    float e1 = exp(pp - p);
    float e2 = exp(ww - p);
    y[ii] = __float2half((e1 * aa + e2 * vv) / (e1 * bb + e2));
    ww = w + pp;
    p = max(ww, kk);
    e1 = exp(ww - p);
    e2 = exp(kk - p);
    aa = e1 * aa + e2 * vv;
    bb = e1 * bb + e2;
    pp = p;
  }
  _aa[_state_offset] = aa;
  _bb[_state_offset] = bb;
  _pp[_state_offset] = pp;
}

void cuda_wkv_forward_new(int B, int T, int C, float *w, float *u, float *k,
                          float *v, half *r, half *y, float *aa, float *bb,
                          float *pp) {
  dim3 threadsPerBlock(min(C, 32));
  assert(B * C % threadsPerBlock.x == 0);
  dim3 numBlocks(B * C / threadsPerBlock.x);
  kernel_wkv_forward_new<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, r,
                                                         y, aa, bb, pp);
}

__global__ void _att_mix(const half *xx, const half *sx, const half *k_mix,
                        const half *v_mix, const half *r_mix,
                        const int outer_size, const int inner_size, half *kx,
                        half *vx, half *rx) {
  for (int idx2 = blockIdx.x * blockDim.x + threadIdx.x; idx2 < inner_size;
       idx2 += blockDim.x * gridDim.x) {
    half k_mix_ = k_mix[idx2];
    half v_mix_ = v_mix[idx2];
    half r_mix_ = r_mix[idx2];
    for (int row = 0; row < outer_size; ++row) {
      int idx1 = row * inner_size + idx2;
      half xx_ = xx[idx1];
      half sx_ = sx[idx1];
      kx[idx1] = __hadd(__hmul(xx_, k_mix_),
                        __hmul(sx_, __hsub(__float2half(1), k_mix_)));
      vx[idx1] = __hadd(__hmul(xx_, v_mix_),
                        __hmul(sx_, __hsub(__float2half(1), v_mix_)));
      rx[idx1] = __hadd(__hmul(xx_, r_mix_),
                        __hmul(sx_, __hsub(__float2half(1), r_mix_)));
    }
  }
}

void att_mix(const half *xx, const half *sx, const half *k_mix,
            const half *v_mix, const half *r_mix, const int outer_size,
            const int inner_size, half *kx, half *vx, half *rx) {
  // 256 is good enough on most GPUs
  const int32_t BLOCK_SIZE = 256;
  assert(inner_size % BLOCK_SIZE == 0);
  _att_mix<<<inner_size / BLOCK_SIZE, BLOCK_SIZE>>>(
      xx, sx, k_mix, v_mix, r_mix, outer_size, inner_size, kx, vx, rx);
}

struct InplaceSigmoid {
  __device__ __forceinline__ half operator()(int i) const {
    ptr[i] = __float2half(1.0 / (1.0 + exp(-__half2float(ptr[i]))));
  }
  half *ptr;
};

struct InplaceMul {
  __device__ __forceinline__ half operator()(int i) const {
    y[i] = __hmul(x[i], y[i]);
  }
  half *y;
  half *x;
};

/*
   Equivalent Python code:

   xx = F.layer_norm(x, (x.shape[-1],), weight=ln_w, bias=ln_b)
   sx = torch.cat((sx.unsqueeze(0), xx[:-1,:]))
   kx = xx * k_mix + sx * (1 - k_mix)
   vx = xx * v_mix + sx * (1 - v_mix)
   rx = xx * r_mix + sx * (1 - r_mix)

   r = torch.sigmoid(gemm(rx, rw))
   k = gemm(kx, kw, output_dtype=torch.float32)
   v = gemm(vx, vw, output_dtype=torch.float32)

   T = x.shape[0]
   for t in range(T):
       kk = k[t]
       vv = v[t]
       ww = t_first + kk
       p = torch.maximum(pp, ww)
       e1 = torch.exp(pp - p)
       e2 = torch.exp(ww - p)
       sx[t] = ((e1 * aa + e2 * vv) / (e1 * bb + e2)).to(dtype=x.dtype)
       ww = t_decay + pp
       p = torch.maximum(ww, kk)
       e1 = torch.exp(ww - p)
       e2 = torch.exp(kk - p)
       aa = e1 * aa + e2 * vv
       bb = e1 * bb + e2
       pp = p
   out = gemm(r * sx, ow)
   return x + out, xx[-1,:], aa, bb, pp
*/
Tensor att_seq(Tensor x, Tensor sx, Tensor ln_w, Tensor ln_b, Tensor k_mix,
               Tensor v_mix, Tensor r_mix, Tensor kw, Tensor vw, Tensor rw,
               Tensor ow, Tensor t_first, Tensor pp, Tensor aa, Tensor bb,
               Tensor t_decay, /* imm */ Tensor buf, /* out */ Tensor x_plus_out) {
  Tensor xx = at::layer_norm(x, {x.size(-1)}, ln_w, ln_b);
  sx = at::cat({sx.unsqueeze(0), xx.slice(0, 0, -1)}, 0);
  char* buf_ptr = (char*)buf.data_ptr();
  half* kx = (half*)buf_ptr;
  half* vx = kx + x.numel();
  half* rx = vx + x.numel();
  half* wkv_y = rx + x.numel();
  att_mix(data_ptr<half>(xx), data_ptr<half>(sx), data_ptr<half>(k_mix),
         data_ptr<half>(v_mix), data_ptr<half>(r_mix), xx.size(0), xx.size(1),
         kx, vx, rx);
  float* k = reinterpret_cast<float*>(wkv_y + x.numel());
  float* v = k + x.size(0) * kw.size(1);
  half* r = reinterpret_cast<half*>(v + x.size(0) * vw.size(1));

  gemm_fp16_cublas(kx, kw.data_ptr(), k, x.size(0), kw.size(1), kw.size(0), true);
  gemm_fp16_cublas(vx, vw.data_ptr(), v, x.size(0), vw.size(1), vw.size(0), true);
  gemm_fp16_cublas(rx, rw.data_ptr(), r, x.size(0), rw.size(1), rw.size(0), false);
  element_wise(InplaceSigmoid{r}, x.size(0) * rw.size(1));
  cuda_wkv_forward_new(1, x.size(0), x.size(1), data_ptr<float>(t_decay),
                       data_ptr<float>(t_first), k, v, r,
                       wkv_y, data_ptr<float>(aa),
                       data_ptr<float>(bb), data_ptr<float>(pp));
  element_wise(InplaceMul{wkv_y, r}, x.numel());
  gemm_fp16_cublas(wkv_y, ow.data_ptr(), x_plus_out.data_ptr(), x.size(0), ow.size(1), ow.size(0), false);
  x_plus_out += x;
  return xx;
}
