#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "element_wise.h"
#include "util.h"

using torch::Tensor;

void gemm_fp16_cublas(const void *a, const void *b, void *c, int ori_m,
                      int ori_n, int ori_k, bool output_fp32);

__global__ void _ffn_seq_mix(const half *xx, const half *sx, const half *k_mix,
                             const half *r_mix, const int outer_size,
                             const int inner_size, half *kx, half *rx) {
  for (int idx2 = blockIdx.x * blockDim.x + threadIdx.x; idx2 < inner_size;
       idx2 += blockDim.x * gridDim.x) {
    half k_mix_ = k_mix[idx2];
    half r_mix_ = r_mix[idx2];
    for (int row = 0; row < outer_size; ++row) {
      int idx1 = row * inner_size + idx2;
      half xx_ = xx[idx1];
      half sx_ = sx[idx1];
      kx[idx1] = __hadd(__hmul(xx_, k_mix_),
                        __hmul(sx_, __hsub(__float2half(1), k_mix_)));
      rx[idx1] = __hadd(__hmul(xx_, r_mix_),
                        __hmul(sx_, __hsub(__float2half(1), r_mix_)));
    }
  }
}

void ffn_seq_mix(const half *xx, const half *sx, const half *k_mix,
                 const half *r_mix, const int outer_size, const int inner_size,
                 half *kx, half *rx) {
  // 256 is good enough on most GPUs
  const int32_t BLOCK_SIZE = 256;
  assert(inner_size % BLOCK_SIZE == 0);
  _ffn_seq_mix<<<inner_size / BLOCK_SIZE, BLOCK_SIZE>>>(
      xx, sx, k_mix, r_mix, outer_size, inner_size, kx, rx);
}

struct InplaceSigmoid {
  __device__ __forceinline__ void operator()(int i) const {
    ptr[i] = __float2half(1.0 / (1.0 + exp(-__half2float(ptr[i]))));
  }
  half *ptr;
};

struct InplaceReLUAndSquare {
  __device__ __forceinline__ void operator()(int i) const {
    // __hmax is not defined in old cuda
    if (__hgt(ptr[i], __float2half(0))) {
      ptr[i] = __hmul(ptr[i], ptr[i]);
    } else {
      ptr[i] = __float2half(0);
    }
  }
  half *ptr;
};

struct InplaceFma {
  __device__ __forceinline__ void operator()(int i) const {
    a[i] = __hfma(a[i], b[i], c[i]);
  }
  half *a;
  const half *b;
  const half *c;
};

/*
   Equivalent Python code:

   xx = F.layer_norm(x, (x.shape[-1],), weight=ln_w, bias=ln_b)
   sx = torch.cat((sx.unsqueeze(0), xx[:-1,:]))
   kx = xx * k_mix + sx * (1 - k_mix)
   rx = xx * r_mix + sx * (1 - r_mix)

   r = torch.sigmoid(gemm(rx, rw))
   vx = torch.square(torch.relu(gemm(kx, kw)))
   out = r * gemm(vx, vw)
   return x + out, xx[-1,:]
*/
Tensor ffn_seq(Tensor x, Tensor sx, Tensor ln_w, Tensor ln_b, Tensor k_mix,
               Tensor r_mix, Tensor kw, Tensor vw, Tensor rw,
               /* imm */ Tensor buf,
               /* out */ Tensor x_plus_out) {
  Tensor xx = at::layer_norm(x, {x.size(-1)}, ln_w, ln_b);
  sx = at::cat({sx.unsqueeze(0), xx.slice(0, 0, -1)}, 0);
  char *buf_ptr = (char *)buf.data_ptr();
  half *kx = (half *)buf_ptr;
  half *rx = kx + x.numel();
  half *vx = rx + x.numel();
  half *r = vx + x.size(0) * kw.size(1);
  ffn_seq_mix(data_ptr<half>(xx), data_ptr<half>(sx), data_ptr<half>(k_mix),
              data_ptr<half>(r_mix), xx.size(0), xx.size(1), kx, rx);

  gemm_fp16_cublas(rx, rw.data_ptr(), r, x.size(0), rw.size(1), x.size(1),
                   false);
  element_wise(InplaceSigmoid{r}, x.size(0) * rw.size(1));
  gemm_fp16_cublas(kx, kw.data_ptr(), vx, x.size(0), kw.size(1), x.size(1),
                   false);
  element_wise(InplaceReLUAndSquare{vx}, x.size(0) * kw.size(1));
  gemm_fp16_cublas(vx, vw.data_ptr(), x_plus_out.data_ptr(), x.size(0),
                   vw.size(1), vw.size(0), false);
  element_wise(InplaceFma{data_ptr<half>(x_plus_out), r, data_ptr<half>(x)},
               x_plus_out.numel());
  return xx;
}

struct FfnOneMix {
  __device__ __forceinline__ void operator()(int idx) {
    half k_mix_ = k_mix[idx];
    half r_mix_ = r_mix[idx];
    half xx_ = xx[idx];
    half sx_ = sx[idx];
    kx[idx] = __hadd(__hmul(xx_, k_mix_),
                     __hmul(sx_, __hsub(__float2half(1), k_mix_)));
    rx[idx] = __hadd(__hmul(xx_, r_mix_),
                     __hmul(sx_, __hsub(__float2half(1), r_mix_)));
  }
  half *k_mix;
  half *r_mix;
  half *xx;
  half *sx;
  half *kx;
  half *rx;
};

/*
  Equivalent Python code:

  xx = F.layer_norm(x, (x.shape[-1],), weight=ln_w, bias=ln_b)
  kx = xx * k_mix + sx * (1 - k_mix)
  rx = xx * r_mix + sx * (1 - r_mix)

  r = torch.sigmoid(gemm(rx, rw))
  vx = torch.square(torch.relu(gemm(kx, kw)))
  out = r * gemm(vx, vw)
  return x + out, xx
*/
Tensor ffn_one(Tensor x, Tensor sx, Tensor ln_w, Tensor ln_b, Tensor k_mix,
               Tensor r_mix, Tensor kw, Tensor vw, Tensor rw,
               /* imm */ Tensor buf,
               /* out */ Tensor x_plus_out) {
  Tensor xx = at::layer_norm(x, {x.size(-1)}, ln_w, ln_b);
  char *buf_ptr = (char *)buf.data_ptr();
  half *kx = (half *)buf_ptr;
  half *rx = kx + x.numel();
  half *vx = rx + x.numel();
  half *r = vx + x.size(0) * kw.size(1);
  element_wise(FfnOneMix{data_ptr<half>(k_mix), data_ptr<half>(r_mix),
                         data_ptr<half>(xx), data_ptr<half>(sx), kx, rx},
               x.numel());
  // vector * matrix, so m = 1
  gemm_fp16_cublas(rx, rw.data_ptr(), r, 1, rw.size(1), rw.size(0), false);
  element_wise(InplaceSigmoid{r}, rw.size(1));
  gemm_fp16_cublas(kx, kw.data_ptr(), vx, 1, kw.size(1), kw.size(0), false);
  element_wise(InplaceReLUAndSquare{vx}, kw.size(1));
  gemm_fp16_cublas(vx, vw.data_ptr(), x_plus_out.data_ptr(), 1, vw.size(1),
                   vw.size(0), false);
  element_wise(InplaceFma{data_ptr<half>(x_plus_out), r, data_ptr<half>(x)},
               x_plus_out.numel());
  return xx;
}
